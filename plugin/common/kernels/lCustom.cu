#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "kernel.h"

// 默认写的.cu是fp32的，TensorRT在fp16运行模式下，运行到不支持fp16的插件op时，会自动切换到fp32模式，等插件op运行完再切换回来。
template <unsigned nthdsPerCTA>
__launch_bounds__(nthdsPerCTA) __global__
    void pCustomKernel(const int n, const float negativeSlope, const float* input, float* output)
{
    for (int i = blockIdx.x * nthdsPerCTA + threadIdx.x; i < n; i += gridDim.x * nthdsPerCTA)
    {
        output[i] = input[i] > 0 ? input[i] : input[i] * negativeSlope;
    }
}

pluginStatus_t CustomGPU(hipStream_t stream, const int n, const float negativeSlope, const void* input, void* output)
{
    const int BS = 512;
    const int GS = (n + BS - 1) / BS;
    pCustomKernel<BS><<<GS, BS, 0, stream>>>(n, negativeSlope,
                                           (const float*) input,
                                           (float*) output);
    return STATUS_SUCCESS;
}

pluginStatus_t CustomInference(
    hipStream_t stream, const int n, const float negativeSlope, const void* input, void* output)
{
    return CustomGPU(stream, n, negativeSlope, (const float*) input, (float*) output);
}
